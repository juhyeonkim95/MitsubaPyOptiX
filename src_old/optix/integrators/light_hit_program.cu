#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include "optix/common/prd_struct.h"
#include "optix/light/light_pdf.h"
#include "optix/common/helpers.h"
using namespace optix;

rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );
rtDeclareVariable(PerRayData_pathtrace_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(int, lightId, , );

rtDeclareVariable(float3,     emission_color, , );
rtBuffer<LightParameter> lights;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable( float3, shading_normal, attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(int, hitTriIdx,  attribute hitTriIdx, );

rtDeclareVariable(float, hit_dist, rtIntersectionDistance, );
rtDeclareVariable(unsigned int,     sample_type, , );
rtDeclareVariable(unsigned int,     use_mis, , );

RT_PROGRAM void diffuseEmitter()
{
    const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    LightParameter light = lights[lightId];

    float3 normal = world_shading_normal;
    if (light.isTwosided == 1){
	    normal = ffnormal;
	}

    float NdotL = dot(normal, -ray.direction);

    if(prd.depth == 0 || use_mis == 0){
        prd.radiance = NdotL >= 0? emission_color : make_float3(0.f);
    } else {

        float lightPdfArea = pdf_light(hitTriIdx, ray.origin, ray.direction, light);

        // float A = light.area;//length(cross(light.v1, light.v2));
        float lightPdf = (hit_dist * hit_dist) / clamp(NdotL, 1.e-3f, 1.0f) * lightPdfArea;

        float mis_weight = powerHeuristic(prd.scatterPdf, lightPdf);
        prd.radiance = NdotL >= 0 ? mis_weight * emission_color : make_float3(0.f);
    }
    prd.t = hit_dist;
    prd.done = true;
}

RT_PROGRAM void any_hit()
{
	prd_shadow.inShadow = true;
	rtTerminateRay();
}