#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include "optix/common/prd_struct.h"
#include "optix/light/light_parameters.h"


using namespace optix;

rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );
rtDeclareVariable(float3, bg_color, , );
rtBuffer<LightParameter> lights;


RT_PROGRAM void miss()
{
    prd.radiance = bg_color;
    prd.done = true;
    prd.t = 1000;
    prd.isMissed = true;
}

//RT_CALLABLE_PROGRAM float3 get_transformed_buffer(float3 v) {
//
//}

RT_PROGRAM void miss_environment_mapping()
{
    LightParameter light = lights[0];

    float3 ray = prd.direction;

    float4 a = make_float4(ray.x, ray.y, ray.z, 0);
    a = light.transformation.transpose() * a;
    ray = normalize(make_float3(a.x, a.y, a.z));

    float phi = atan2f(ray.x, -ray.z);
    float theta = acosf(-ray.y);
    float u = (phi + M_PIf) * (0.5f * M_1_PIf);
    float v = theta * M_1_PIf;


//    float theta = atan2f( ray.x, ray.z );
//    float phi   = M_PIf * 0.5f -  acosf( ray.y );
//    float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
//    float v     = 0.5f * ( 1.0f + sin(phi) );

    const float3 emission = make_float3(optix::rtTex2D<float4>(light.envmapID , u, v));
    prd.radiance = emission;
    prd.done = true;
    prd.t = 1000;
    prd.isMissed = false;
}